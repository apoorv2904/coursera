#include "hip/hip_runtime.h"
#include    <wb.h>


#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

#define Mask_width  5
#define Mask_radius 2
#define O_TILE_WIDTH 12
#define BLOCK_WIDTH 16

//@@ INSERT CODE HERE
__global__ void convolution_2D_kernel(float* out, float* in, int height,
                                      int width, int channels,
                                      const float *mask) {
    __shared__ float NS[BLOCK_WIDTH][BLOCK_WIDTH][3];   
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row_o = blockIdx.y * O_TILE_WIDTH + ty;
    int col_o = blockIdx.x * O_TILE_WIDTH + tx;
    int row_i = row_o - Mask_radius;
    int col_i = col_o - Mask_radius;
    if (row_i >= 0 && row_i < height && col_i >= 0 && col_i < width) {
        for (int cz = 0; cz < 3; ++cz)
        { NS[ty][tx][cz] = in[(row_i * width + col_i) * channels + cz]; }
    }
    else {
        for (int cz = 0; cz < 3; ++cz) NS[ty][tx][cz] = 0.0;
    }

//  ensure all threads have finished loading data
    __syncthreads();

    if (ty < O_TILE_WIDTH && tx < O_TILE_WIDTH) {
        float res[3];
        for (int i = 0; i < 3; ++i) res[i] = 0.0;
        for (int i = 0; i < Mask_width; ++i) {
            for (int j = 0; j < Mask_width; ++j) {
                for (int ch = 0; ch < 3; ++ch) { res[ch] = res[ch] + mask[i * Mask_width + j] * NS[i + ty][j + tx][ch]; }
            }
        }
        __syncthreads();
        for (int i = 0; i < 3; ++i) {
            if (res[i] < 0.0) res[i] = 0.0;
            if (res[i] > 1.0) res[i] = 1.0;
        }
        if (row_o < height && col_o < width)
            for (int ch = 0; ch < 3; ++ch) { out[(row_o * width + col_o) * channels + ch] = res[ch]; }
    }
    // __syncthreads();
    return;
}

int main(int argc, char* argv[]) {
    wbArg_t args;
    int maskRows;
    int maskColumns;
    int imageChannels;
    int imageWidth;
    int imageHeight;
    char * inputImageFile;
    char * inputMaskFile;
    wbImage_t inputImage;
    wbImage_t outputImage;
    float * hostInputImageData;
    float * hostOutputImageData;
    float * hostMaskData;
    float * deviceInputImageData;
    float * deviceOutputImageData;
    float * deviceMaskData;

    args = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(args, 0);
    inputMaskFile = wbArg_getInputFile(args, 1);

    inputImage = wbImport(inputImageFile);
    hostMaskData = (float *) wbImport(inputMaskFile, &maskRows, &maskColumns);

    assert(maskRows == 5); /* mask height is fixed to 5 in this mp */
    assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);
    assert(imageChannels == 3);

    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

    hostInputImageData = wbImage_getData(inputImage);
    hostOutputImageData = wbImage_getData(outputImage);

    wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

    wbTime_start(GPU, "Doing GPU memory allocation");
    hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * sizeof(float));
    wbTime_stop(GPU, "Doing GPU memory allocation");


    wbTime_start(Copy, "Copying data to the GPU");
    hipMemcpy(deviceInputImageData,
               hostInputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(deviceMaskData,
               hostMaskData,
               maskRows * maskColumns * sizeof(float),
               hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying data to the GPU");


    wbTime_start(Compute, "Doing the computation on the GPU");
    //@@ INSERT CODE HERE
    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
    dim3 dimGrid((imageWidth - 1) / O_TILE_WIDTH + 1, (imageHeight - 1) / O_TILE_WIDTH + 1, 1);
    convolution_2D_kernel<<<dimGrid, dimBlock>>>(deviceOutputImageData, deviceInputImageData,
                                                 imageHeight, imageWidth, imageChannels, deviceMaskData);
    wbTime_stop(Compute, "Doing the computation on the GPU");


    wbTime_start(Copy, "Copying data from the GPU");
    hipMemcpy(hostOutputImageData,
               deviceOutputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying data from the GPU");

    wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

    wbSolution(args, outputImage);

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);

    free(hostMaskData);
    wbImage_delete(outputImage);
    wbImage_delete(inputImage);

    return 0;
}
