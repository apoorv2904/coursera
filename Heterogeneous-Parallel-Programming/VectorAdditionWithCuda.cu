#include "hip/hip_runtime.h"
#include    <wb.h>

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
    int tx = threadIdx.x + blockDim.x * blockIdx.x;
    if (tx < len) out[tx] = in1[tx] + in2[tx];
    return;
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * d11, *d21;
    float * d12, *d22;
    float * dout1, *dout2;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    const int segsize = 4096;
    hipMalloc((void **)&d11, segsize * sizeof(float));
    hipMalloc((void **)&d12, segsize * sizeof(float));
    hipMalloc((void **)&dout1, segsize * sizeof(float));
    hipMalloc((void **)&d21, segsize * sizeof(float));
    hipMalloc((void **)&d22, segsize * sizeof(float));
    hipMalloc((void **)&dout2, segsize * sizeof(float));
    
    hipStream_t s1, s2, s3, s4;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);
    hipStreamCreate(&s3);
    hipStreamCreate(&s4);

    // use one stream currently
    int number[2];
    for (int i = 0; i < inputLength; i += 2 * segsize) {
        // handle boundary conditions in case (inputLength % segsize != 0)
        for (int j = 0; j < 2; ++j) {
            if (i + j * segsize + segsize <= inputLength) number[j] = segsize;
            else if (i + j * segsize < inputLength) number[j] = inputLength - i - j * segsize;
            else number[j] = 0;
        }
        hipMemcpyAsync(d11, hostInput1 + i, number[0] * sizeof(float), hipMemcpyHostToDevice, s1);
        hipMemcpyAsync(d12, hostInput2 + i, number[0] * sizeof(float), hipMemcpyHostToDevice, s1);
        hipMemcpyAsync(d21, hostInput1 + i + segsize * 1, number[1] * sizeof(float), hipMemcpyHostToDevice, s2);
        hipMemcpyAsync(d22, hostInput2 + i + segsize * 1, number[1] * sizeof(float), hipMemcpyHostToDevice, s2);
        vecAdd<<<(number[0] - 1) / 256 + 1, 256, 0, s1>>>(d11, d12, dout1, number[0]);
        hipMemcpyAsync(hostOutput + i, dout1, number[0] * sizeof(float), hipMemcpyDeviceToHost, s1);
        vecAdd<<<(number[1] - 1) / 256 + 1, 256, 0, s2>>>(d21, d22, dout2, number[1]);
        hipMemcpyAsync(hostOutput + i + segsize * 1, dout2, number[1] * sizeof(float), hipMemcpyDeviceToHost, s2);
    }

    hipFree(d11); hipFree(d21);
    hipFree(d12); hipFree(d22);
    hipFree(dout1); hipFree(dout2);

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}

